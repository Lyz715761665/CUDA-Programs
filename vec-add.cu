// Adapted from vec-add-start.cu

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c,int length) { 
  // calculate the unique thread index
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  // perform tid-th elements addition 
  if (tid < length) {
    c[tid] = a[tid] + b[tid];
  }
}

__host__ void usage() {
	fprintf(stderr, "Usage: vec-add size\n");
	exit(1);
}


int main(int argc, char** argv) {
  int *d_a, *d_b, *d_c; // device copies of a, b, c 
  int length; 
  int threadsPerBlock,blocksPerGrid;

  ///Setup input vector size
  if (argc != 2) 
    usage();
  if (sscanf(argv[1], "%d", &length) != 1) 
    usage();
               
  //allocate memory for host vectors in CPU
  int* a = (int*)malloc(sizeof(int)*length);
  int* b = (int*)malloc(sizeof(int)*length);
  int* c = (int*)malloc(sizeof(int)*length);

  //initialize vector contents
  for (int i=0;i<length;i++){
    a[i]=1;
    b[i]=1;
  }

  //Max threads per block for lo0 from DQ
  if (length<1024){
    threadsPerBlock = length;
    blocksPerGrid   = 1;
  } else {
    threadsPerBlock = 1024;
    blocksPerGrid   = ceil(double(length)/double(threadsPerBlock));
  }


  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, sizeof(int)*length); 
  hipMalloc((void **)&d_b, sizeof(int)*length); 
  hipMalloc((void **)&d_c, sizeof(int)*length);

  // Copy inputs to device
  hipMemcpy(d_a, a, sizeof(int)*length, hipMemcpyHostToDevice); 
  hipMemcpy(d_b, b, sizeof(int)*length, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  add<<<blocksPerGrid,threadsPerBlock>>>(d_a, d_b, d_c,length);

  // Copy result back to host
  hipMemcpy(c, d_c, sizeof(int)*length, hipMemcpyDeviceToHost);
  fprintf(stdout, "There are %d 2s in vector c\n", length); 

  // Cleanup
  hipFree(d_a); 
  hipFree(d_b); 
  hipFree(d_c);
  
  free(a);
  free(b);
  free(c);

  return(0);
}
